#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

hipError_t cuda();

__global__ void kernel(){
  
}

__device__
float2 Hash2(float2 p)
{
    float2 value;
    value.x = 53.3158f;
    value.y = 43.6143f;
    float r = 523.0f*sinf(p.x*value.x+ p.y*value.y);
    float v1 = 15.32354 * r - floor(15.32354 * r);
    float v2 = 17.25865 * r - floor(17.25865 * r);
    value.x = v1;
    value.y = v2;
    return value;
}
__device__ float Cells(float2 p, float numCells)
{
    p.x *= numCells;
    p.y *= numCells;
    float d = 1.0e10;
    for (int xo = -1; xo <= 1; xo++)
    {
        for (int yo = -1; yo <= 1; yo++)
        {
            float2 value; 
            value.x = (float)xo;
            value.y = (float)yo;
            float2 tp;
            tp.x = floorf(p.x) + value.x;
            tp.y = floorf(p.y) + value.y;
            float div = (float)(numCells / 2.0);
            float v1 = fmod(tp.x, div);
            float v2 = fmod(tp.y, div);
            float2 v;
            v.x = v1;
            v.y = v2;
            v = Hash2(v);
            tp.x = p.x - tp.x - v.x;
            tp.y = p.y - tp.y - v.y;
            d = min(d, tp.x*tp.x+tp.y*tp.y);
        }
    }
    return sqrt(d);
}

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b)
{
	return max(a, min(b, x));
}

__device__ int clamp(int x, int a, int b)
{
	return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
	r = clamp(r, 0.0f, 255.0f);
	g = clamp(g, 0.0f, 255.0f);
	b = clamp(b, 0.0f, 255.0f);
    return (int(r) << 16) | (int(g) << 8) | int(b);
}

__global__ void
cudaRender(unsigned int *g_odata, int imgw)
{
	extern __shared__ uchar4 sdata[];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x*bw + tx;
	int y = blockIdx.y*bh + ty;

    float2 uv;
    uv.x = (float)x / 800.0;
    uv.y = (float)y / 800.0;

    float c = Cells(uv, 16.0);

	uchar4 c4 = make_uchar4(c*0.83*255.0, c*255.0, min(c*1.3, 1.0)*255.0, 0);
	g_odata[y*imgw + x] = rgbToInt(c4.z, c4.y, c4.x);
}

extern "C" void
launch_cudaRender(dim3 grid, dim3 block, int sbytes, unsigned int *g_odata, int imgw)
{
	cudaRender << < grid, block, sbytes >> >(g_odata, imgw);
}
